#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>

#include "layer_params.cuh"

void ConvLayerParams::initializeValues(
    hipdnnHandle_t cudnn_handle, ConvDescriptor *user_params,
    hipdnnDataType_t data_type, int batch_size,
    hipdnnTensorFormat_t tensor_format, size_t data_type_size,
    LayerDimension &output_size, UpdateRule update_rule)
{
  // create tensor, filter, conv descriptor
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));

  C_in = user_params->input_channels;
  C_out = user_params->output_channels;
  filter_h = user_params->kernel_h;
  filter_w = user_params->kernel_w;
  kernel_size = C_out * C_in * filter_h * filter_w;
  this->data_type = data_type;
  this->activation_mode = user_params->activation_mode;

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      input_tensor, tensor_format, data_type, batch_size,
      user_params->input_channels, user_params->input_h, user_params->input_w));

  checkCUDNN(hipdnnSetFilter4dDescriptor(
      filter_desc, data_type, tensor_format, user_params->output_channels,
      user_params->input_channels, user_params->kernel_h,
      user_params->kernel_w));

  int dilation_h = 1, dilation_w = 1;
  checkCUDNN(hipdnnSetConvolution2dDescriptor(
      conv_desc, user_params->pad_h, user_params->pad_w, user_params->stride_y,
      user_params->stride_x, dilation_h, dilation_w, HIPDNN_CROSS_CORRELATION,
      data_type));

  int output_batch_size, output_channels, output_h, output_w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(
      conv_desc, input_tensor, filter_desc, &output_batch_size,
      &output_channels, &output_h, &output_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(output_tensor, tensor_format, data_type,
                                        output_batch_size, output_channels,
                                        output_h, output_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(bias_desc, tensor_format, data_type, 1,
                                        output_channels, 1, 1));

  fwd_req_count = 10;
  fwd_perf = (hipdnnConvolutionFwdAlgoPerf_t *)malloc(
      fwd_req_count * sizeof(hipdnnConvolutionFwdAlgoPerf_t));
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
      cudnn_handle, input_tensor, filter_desc, conv_desc, output_tensor,
      fwd_req_count, &fwd_ret_count, fwd_perf));

  // std::cout << "Printing forward conv algo perf\n";
  // std::cout << "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM: " <<
  // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM << std::endl; for (int i = 0; i <
  // fwd_ret_count; i++) { 	std::cout << i << std::endl; 	std::cout << "algo:
  // " << fwd_perf[i].algo << std::endl; 	std::cout << "status: " <<
  // hipdnnGetErrorString(fwd_perf[i].status) << std::endl; 	std::cout <<
  // "time(ms): " << fwd_perf[i].time << std::endl; 	std::cout <<
  // "memory(MB):
  // "
  // << fwd_perf[i].memory * 1.0 / 1024 / 1024 << std::endl; 	std::cout <<
  // "mathType: " << fwd_perf[i].mathType << std::endl; 	std::cout <<
  // std::endl;
  // }

  bwd_filter_req_count = 10;
  bwd_filter_perf = (hipdnnConvolutionBwdFilterAlgoPerf_t *)malloc(
      bwd_filter_req_count * sizeof(hipdnnConvolutionBwdFilterAlgoPerf_t));
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithm(
      cudnn_handle, input_tensor, output_tensor, conv_desc, filter_desc,
      bwd_filter_req_count, &bwd_filter_ret_count, bwd_filter_perf));

  // std::cout << "Printing bwdfilter conv algo perf\n";
  // std::cout << "HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1 " <<
  // HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1 << std::endl; for (int i = 0; i <
  // bwd_filter_ret_count; i++) { 	std::cout << i << std::endl; 	std::cout <<
  // "algo: " << bwd_filter_perf[i].algo << std::endl; 	std::cout << "status: "
  // << hipdnnGetErrorString(bwd_filter_perf[i].status) << std::endl; std::cout
  // << "time(ms): " << bwd_filter_perf[i].time << std::endl; 	std::cout <<
  // "memory(MB): " << bwd_filter_perf[i].memory * 1.0 / 1024 / 1024 <<
  // std::endl; 	std::cout << "mathType: " << bwd_filter_perf[i].mathType <<
  // std::endl; 	std::cout << std::endl;
  // }
  bwd_data_req_count = 10;
  bwd_data_perf = (hipdnnConvolutionBwdDataAlgoPerf_t *)malloc(
      bwd_data_req_count * sizeof(hipdnnConvolutionBwdDataAlgoPerf_t));
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithm(
      cudnn_handle, filter_desc, output_tensor, conv_desc, input_tensor,
      bwd_data_req_count, &bwd_data_ret_count, bwd_data_perf));

  // std::cout << "Printing bwddata conv algo perf\n";
  // for (int i = 0; i < bwd_data_ret_count; i++) {
  // 	std::cout << i << std::endl;
  // 	std::cout << "algo: " << bwd_data_perf[i].algo << std::endl;
  // 	std::cout << "status: " << hipdnnGetErrorString(bwd_data_perf[i].status)
  // << std::endl; 	std::cout << "time(ms): " << bwd_data_perf[i].time <<
  // std::endl; 	std::cout << "memory(MB): " << bwd_data_perf[i].memory * 1.0
  // / 1024 / 1024 << std::endl; 	std::cout << "mathType: " <<
  // bwd_data_perf[i].mathType << std::endl; 	std::cout << std::endl;
  // }

  this->update_rule = update_rule;

  hipdnnActivationMode_t mode;
  if (activation_mode == SIGMOID)
    mode = HIPDNN_ACTIVATION_SIGMOID;
  else if (activation_mode == RELU)
    mode = HIPDNN_ACTIVATION_RELU;
  else if (activation_mode == TANH)
    mode = HIPDNN_ACTIVATION_TANH;
  else if (activation_mode == CLIPPED_RELU)
    mode = HIPDNN_ACTIVATION_CLIPPED_RELU;
  else if (activation_mode == ELU)
    mode = HIPDNN_ACTIVATION_ELU;

  if (activation_mode != ACTIVATION_NONE)
  {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actv_desc));
    checkCUDNN(hipdnnSetActivationDescriptor(
        actv_desc, mode, HIPDNN_PROPAGATE_NAN, user_params->actv_coef));
  }

  output_size.N = output_batch_size, output_size.C = output_channels,
  output_size.H = output_h, output_size.W = output_w;
}

void ConvLayerParams::allocateSpace(hiprandGenerator_t curand_gen,
                                    hipdnnDataType_t data_type,
                                    size_t data_type_size, float std_dev,
                                    size_t &free_bytes)
{
  if (kernel_size % 2 != 0)
    kernel_size += 1;
  hipMallocManaged(&W, kernel_size * data_type_size);
  hipMallocManaged(&b, C_out * data_type_size);

  hipMallocManaged(&dW, kernel_size * data_type_size);
  hipMallocManaged(&db, C_out * data_type_size);

  if (data_type == HIPDNN_DATA_FLOAT)
  {
    hiprandGenerateNormal(curand_gen, (float *)W, kernel_size, 0, std_dev);
    fillValue<float><<<ceil(1.0 * C_out / BW), BW>>>((float *)b, C_out, 0);
  }
  else
  {
    hiprandGenerateNormalDouble(curand_gen, (double *)W, kernel_size, 0,
                               std_dev);
    fillValue<double><<<ceil(1.0 * C_out / BW), BW>>>((double *)b, C_out, 0);
  }

  free_bytes = free_bytes - 2 * (kernel_size + C_out) * data_type_size;
}

void ConvLayerParams::stepParams(hipblasHandle_t cublas_handle,
                                 double learning_rate)
{
  float Salpha = -learning_rate;
  double Dalpha = -learning_rate;

  if (update_rule == SGD)
  {
    if (data_type == HIPDNN_DATA_FLOAT)
    {
      hipblasSaxpy(cublas_handle, kernel_size, &Salpha, (float *)dW, 1,
                  (float *)W, 1);

      hipblasSaxpy(cublas_handle, C_out, &Salpha, (float *)db, 1, (float *)b, 1);
    }
    else if (data_type == HIPDNN_DATA_DOUBLE)
    {
      hipblasDaxpy(cublas_handle, kernel_size, &Dalpha, (double *)dW, 1,
                  (double *)W, 1);

      hipblasDaxpy(cublas_handle, C_out, &Dalpha, (double *)db, 1, (double *)b,
                  1);
    }
  }
}

size_t ConvLayerParams::getWorkspaceSize(
    size_t &free_bytes, ConvLayerParams::ConvDirection conv_direction)
{
  if (conv_direction == FWD)
  {
    if (fwd_perf[0].memory > free_bytes)
      outOfMemory();
    fwd_algo = fwd_perf[0].algo;
    return fwd_perf[0].memory;
  }
  else if (conv_direction == BWD_FILTER)
  {
    if (bwd_filter_perf[0].memory > free_bytes)
      outOfMemory();
    bwd_filter_algo = bwd_filter_perf[0].algo;
    return bwd_filter_perf[0].memory;
  }
  else if (conv_direction == BWD_DATA)
  {
    if (bwd_data_perf[0].memory > free_bytes)
      outOfMemory();
    bwd_data_algo = bwd_data_perf[0].algo;
    return bwd_data_perf[0].memory;
  }
  return 0;
}

void FCLayerParams::initializeValues(FCDescriptor *user_params, int batch_size,
                                     hipdnnTensorFormat_t tensor_format,
                                     hipdnnDataType_t data_type,
                                     LayerDimension &output_size,
                                     UpdateRule update_rule)
{
  C_in = user_params->input_channels;
  C_out = user_params->output_channels;
  weight_matrix_size = C_in * C_out;
  this->data_type = data_type;
  this->activation_mode = user_params->activation_mode;

  this->update_rule = update_rule;

  hipdnnActivationMode_t mode;
  if (activation_mode == SIGMOID)
    mode = HIPDNN_ACTIVATION_SIGMOID;
  else if (activation_mode == RELU)
    mode = HIPDNN_ACTIVATION_RELU;
  else if (activation_mode == TANH)
    mode = HIPDNN_ACTIVATION_TANH;
  else if (activation_mode == CLIPPED_RELU)
    mode = HIPDNN_ACTIVATION_CLIPPED_RELU;
  else if (activation_mode == ELU)
    mode = HIPDNN_ACTIVATION_ELU;

  if (activation_mode != ACTIVATION_NONE)
  {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actv_desc));
    checkCUDNN(hipdnnSetActivationDescriptor(
        actv_desc, mode, HIPDNN_PROPAGATE_NAN, user_params->actv_coef));
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_tensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_tensor, tensor_format,
                                          data_type, batch_size,
                                          user_params->output_channels, 1, 1));
  }

  output_size.N = batch_size, output_size.C = C_out,
  output_size.H = output_size.W = 1;
}

void FCLayerParams::allocateSpace(hiprandGenerator_t curand_gen,
                                  hipdnnDataType_t data_type,
                                  size_t data_type_size, float std_dev,
                                  size_t &free_bytes)
{
  int wt_alloc_size = weight_matrix_size;
  if (wt_alloc_size % 2 != 0)
    wt_alloc_size += 1;
  hipMallocManaged(&W, wt_alloc_size * data_type_size);
  hipMallocManaged(&b, C_out * data_type_size);

  hipMallocManaged(&dW, wt_alloc_size * data_type_size);
  hipMallocManaged(&db, C_out * data_type_size);

  if (data_type == HIPDNN_DATA_FLOAT)
  {
    hiprandGenerateNormal(curand_gen, (float *)W, wt_alloc_size, 0, std_dev);
    fillValue<float><<<ceil(1.0 * C_out / BW), BW>>>((float *)b, C_out, 0);
  }
  else if (data_type == HIPDNN_DATA_DOUBLE)
  {
    hiprandGenerateNormalDouble(curand_gen, (double *)W, wt_alloc_size, 0,
                               std_dev);
    fillValue<double><<<ceil(1.0 * C_out / BW), BW>>>((double *)b, C_out, 0);
  }
  free_bytes = free_bytes - 2 * (C_in * C_out + C_out) * data_type_size;
}

void FCLayerParams::stepParams(hipblasHandle_t cublas_handle,
                               double learning_rate)
{
  float Salpha = -learning_rate;
  double Dalpha = -learning_rate;

  // {
  // 	float *db_h = (float *)malloc(C_out * sizeof(float));
  // 	checkCudaErrors(hipMemcpy(db_h, db, C_out * sizeof(float),
  // hipMemcpyDeviceToHost)); 	for (int i = 0; i < C_out; i++) { std::cout <<
  // db_h[i] << ' ';
  // 	}
  // 	std::cout << "\n";
  // 	int n;
  // 	std::cin >> n;
  // }

  if (update_rule == SGD)
  {
    if (data_type == HIPDNN_DATA_FLOAT)
    {
      hipblasSaxpy(cublas_handle, weight_matrix_size, &Salpha, (float *)dW, 1,
                  (float *)W, 1);

      hipblasSaxpy(cublas_handle, C_out, &Salpha, (float *)db, 1, (float *)b, 1);
    }
    else if (data_type == HIPDNN_DATA_DOUBLE)
    {
      hipblasDaxpy(cublas_handle, weight_matrix_size, &Dalpha, (double *)dW, 1,
                  (double *)W, 1);

      hipblasDaxpy(cublas_handle, C_out, &Dalpha, (double *)db, 1, (double *)b,
                  1);
    }
  }
  // {
  // 	float *db_h = (float *)malloc(C_out * sizeof(float));
  // 	checkCudaErrors(hipMemcpy(db_h, b, C_out * sizeof(float),
  // hipMemcpyDeviceToHost)); 	for (int i = 0; i < C_out; i++) { std::cout <<
  // db_h[i] << ' ';
  // 	}
  // 	std::cout << "\n";
  // 	int n;
  // 	std::cin >> n;
  // }
}

void PoolingLayerParams::initializeValues(PoolingDescriptor *user_params,
                                          hipdnnDataType_t data_type,
                                          hipdnnTensorFormat_t tensor_format,
                                          int batch_size,
                                          LayerDimension &output_size)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_tensor));

  checkCUDNN(hipdnnSetTensor4dDescriptor(
      input_tensor, tensor_format, data_type, batch_size,
      user_params->input_channels, user_params->input_h, user_params->input_w));

  checkCUDNN(hipdnnCreatePoolingDescriptor(&pool_desc));

  hipdnnPoolingMode_t mode;
  if (user_params->mode == POOLING_MAX)
    mode = HIPDNN_POOLING_MAX;
  else if (user_params->mode == POOLING_AVERAGE_COUNT_INCLUDE_PADDING)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
  else if (user_params->mode == POOLING_AVERAGE_COUNT_EXCLUDE_PADDING)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;

  checkCUDNN(hipdnnSetPooling2dDescriptor(
      pool_desc, mode, HIPDNN_PROPAGATE_NAN, user_params->kernel_h,
      user_params->kernel_w, user_params->pad_h, user_params->pad_w,
      user_params->stride_y, user_params->stride_x));

  int output_batch_size, output_channels, output_h, output_w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(
      pool_desc, input_tensor, &output_batch_size, &output_channels, &output_h,
      &output_w));

  checkCUDNN(hipdnnSetTensor4dDescriptor(output_tensor, tensor_format, data_type,
                                        output_batch_size, output_channels,
                                        output_h, output_w));

  output_size.N = output_batch_size, output_size.C = output_channels,
  output_size.H = output_h, output_size.W = output_w;
}

void PoolingLayerParams::allocateSpace(size_t &free_bytes)
{
}

void ActivationLayerParams::initializeValues(ActivationDescriptor *user_params,
                                             hipdnnDataType_t data_type,
                                             hipdnnTensorFormat_t tensor_format,
                                             int batch_size,
                                             LayerDimension &output_size)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));

  checkCUDNN(hipdnnSetTensor4dDescriptor(input_tensor, tensor_format, data_type,
                                        batch_size, user_params->channels,
                                        user_params->h, user_params->w));

  hipdnnActivationMode_t mode;
  if (user_params->mode == SIGMOID)
    mode = HIPDNN_ACTIVATION_SIGMOID;
  else if (user_params->mode == RELU)
    mode = HIPDNN_ACTIVATION_RELU;
  else if (user_params->mode == TANH)
    mode = HIPDNN_ACTIVATION_TANH;
  else if (user_params->mode == CLIPPED_RELU)
    mode = HIPDNN_ACTIVATION_CLIPPED_RELU;
  else if (user_params->mode == ELU)
    mode = HIPDNN_ACTIVATION_ELU;

  checkCUDNN(hipdnnCreateActivationDescriptor(&actv_desc));
  checkCUDNN(hipdnnSetActivationDescriptor(actv_desc, mode, HIPDNN_PROPAGATE_NAN,
                                          user_params->coef));

  output_size.N = batch_size, output_size.C = user_params->channels,
  output_size.H = user_params->h, output_size.W = user_params->w;
}

void ActivationLayerParams::allocateSpace(size_t &free_bytes)
{
}

void SoftmaxLayerParams::initializeValues(SoftmaxDescriptor *user_params,
                                          hipdnnDataType_t data_type,
                                          hipdnnTensorFormat_t tensor_format,
                                          int batch_size,
                                          LayerDimension &output_size)
{
  if (user_params->algo == SOFTMAX_FAST)
    algo = HIPDNN_SOFTMAX_FAST;
  else if (user_params->algo == SOFTMAX_ACCURATE)
    algo = HIPDNN_SOFTMAX_ACCURATE;

  if (user_params->mode == SOFTMAX_MODE_INSTANCE)
    mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
  else if (user_params->mode == SOFTMAX_MODE_CHANNEL)
  {
    mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
  }

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_tensor, tensor_format, data_type,
                                        batch_size, user_params->channels,
                                        user_params->h, user_params->w));

  output_size.N = batch_size, output_size.C = user_params->channels,
  output_size.H = user_params->h, output_size.W = user_params->w;
}

void SoftmaxLayerParams::allocateSpace(size_t &free_bytes)
{
}