#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <time.h>

#include <cstdio>
#include <string>

#include "neural_net.cuh"

template <typename T>
__global__ void inferClass(T *O, int *pred_y, int batch_size, int num_classes)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= batch_size)
    return;

  T max = O[i * num_classes];
  int index = 0;
  for (int j = 1; j < num_classes; j++)
  {
    if (O[i * num_classes + j] > max)
    {
      max = O[i * num_classes + j];
      index = j;
    }
  }
  pred_y[i] = index;
}

void NeuralNet::compareOutputCorrect(int *correct_count, int *y)
{
  *correct_count = 0;

  if (data_type == HIPDNN_DATA_FLOAT)
  {
    float *typecast_O = (float *)layer_input[num_layers - 1];
    inferClass<float><<<ceil(1.0 * batch_size / BW), BW>>>(
        typecast_O, pred_y, batch_size, num_classes);
    for (int i = 0; i < batch_size; i++)
    {
      cout << "Predicted: " << pred_y[i] << " Actual: " << y[i] << "\n";
      if (pred_y[i] == y[i])
        *correct_count = *correct_count + 1;
    }
  }
  else if (data_type == HIPDNN_DATA_DOUBLE)
  {
    double *typecast_O = (double *)layer_input[num_layers - 1];
    inferClass<double><<<ceil(1.0 * batch_size / BW), BW>>>(
        typecast_O, pred_y, batch_size, num_classes);
    for (int i = 0; i < batch_size; i++)
    {
      cout << "Predicted: " << pred_y[i] << " Actual: " << y[i] << "\n";
      if (pred_y[i] == y[i])
        *correct_count = *correct_count + 1;
    }
  }
}
