#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <time.h>

#include <cstdio>
#include <string>

#include "neural_net.cuh"

NeuralNet::NeuralNet() {
  this->num_layers = 0;
  this->batch_size = 0;
}

NeuralNet::NeuralNet(std::vector<LayerSpecifier> &layers, DataType data_type, int batch_size,
                     TensorFormat tensor_format, float softmax_eps, float init_std_dev,
                     UpdateRule update_rule) {
  hipStreamCreate(&stream_compute);
  hipStreamCreate(&stream_memory);

  // create handle
  checkCUDNN(hipdnnCreate(&cudnn_handle));
  checkCUDNN(hipdnnSetStream(cudnn_handle, stream_compute));

  hipblasCreate(&cublas_handle);
  hipblasSetStream(cublas_handle, stream_compute);

  hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetStream(curand_gen, stream_compute);

  hipMemGetInfo(&free_bytes, &total_bytes);
  init_free_bytes = free_bytes;

  if (data_type == DATA_FLOAT) {
    this->data_type = HIPDNN_DATA_FLOAT;
    data_type_size = sizeof(float);
  } else if (data_type == DATA_DOUBLE) {
    this->data_type = HIPDNN_DATA_DOUBLE;
    data_type_size = sizeof(double);
  }

  if (tensor_format == TENSOR_NCHW) {
    this->tensor_format = HIPDNN_TENSOR_NCHW;
  } else if (tensor_format == TENSOR_NHWC) {
    this->tensor_format = HIPDNN_TENSOR_NHWC;
  }

  this->batch_size = batch_size;
  this->softmax_eps = softmax_eps;
  this->init_std_dev = init_std_dev;

  init_layers(layers, update_rule);

  hipMemGetInfo(&free_bytes, &total_bytes);
  std::cout << "Free gigabytes just before allocate space: "
            << free_bytes / (1024.0 * 1024.0 * 1024.0) << std::endl;

  allocate_mem_for_layers(layers);
  hipDeviceSynchronize();
  hipMemGetInfo(&free_bytes, &total_bytes);
  std::cout << "Free gigabytes just after allocate space: "
            << free_bytes / (1024.0 * 1024.0 * 1024.0) << std::endl;

  // Very small - could be allocated initially itself
  hipMallocManaged((void **)&y, batch_size * sizeof(int));
  hipMallocManaged((void **)&pred_y, batch_size * sizeof(int));
  hipMallocManaged((void **)&loss, batch_size * sizeof(float));
  hipMallocManaged(&one_vec, batch_size * data_type_size);

  if (this->data_type == HIPDNN_DATA_FLOAT) {
    fillValue<float><<<ceil(1.0 * batch_size / BW), BW>>>((float *)one_vec, batch_size, 1);
  } else {
    fillValue<double><<<ceil(1.0 * batch_size / BW), BW>>>((double *)one_vec, batch_size, 1);
  }

  allocate_workspace_for_layers(layers);

  hipDeviceSynchronize();
  hipMemGetInfo(&free_bytes, &total_bytes);

  hipDeviceSynchronize();

  size_t temp_free_bytes;
  hipMemGetInfo(&temp_free_bytes, &total_bytes);
  std::cout << "Free gigabytes just before end of NeuralNet: "
            << temp_free_bytes / (1024.0 * 1024.0 * 1024.0) << std::endl;
}

void NeuralNet::init_layers(std::vector<LayerSpecifier> &layers, UpdateRule update_rule) {
  num_layers = layers.size();

  // Allocation of space for input to each layer
  layer_input = (void **)malloc((num_layers + 1) * sizeof(void *));
  layer_input_size = (int *)malloc((num_layers + 1) * sizeof(int));
  dlayer_input = (void **)malloc((num_layers + 1) * sizeof(void *));
  params = (void **)malloc(num_layers * sizeof(void *));

  LayerDimension current_output_size;
  for (int i = 0; i < num_layers; i++) {
    layer_type.push_back(layers[i].type);
    if (layers[i].type == CONV) {
      ConvDescriptor *user_params = (ConvDescriptor *)layers[i].params;
      params[i] = malloc(sizeof(ConvLayerParams));
      ((ConvLayerParams *)params[i])
          ->initializeValues(cudnn_handle, user_params, this->data_type, batch_size,
                             this->tensor_format, data_type_size, current_output_size, update_rule);
    } else if (layers[i].type == FULLY_CONNECTED) {
      FCDescriptor *user_params = (FCDescriptor *)layers[i].params;
      params[i] = malloc(sizeof(FCLayerParams));
      ((FCLayerParams *)params[i])
          ->initializeValues(user_params, batch_size, this->tensor_format, this->data_type,
                             current_output_size, update_rule);
    } else if (layers[i].type == POOLING) {
      PoolingDescriptor *user_params = (PoolingDescriptor *)layers[i].params;
      params[i] = malloc(sizeof(PoolingLayerParams));
      ((PoolingLayerParams *)params[i])
          ->initializeValues(user_params, this->data_type, this->tensor_format, batch_size,
                             current_output_size);
    } else if (layers[i].type == ACTV) {
      ActivationDescriptor *user_params = (ActivationDescriptor *)layers[i].params;
      params[i] = malloc(sizeof(ActivationLayerParams));
      ((ActivationLayerParams *)params[i])
          ->initializeValues(user_params, this->data_type, this->tensor_format, batch_size,
                             current_output_size);
    }

    else if (layers[i].type == SOFTMAX) {
      SoftmaxDescriptor *user_params = (SoftmaxDescriptor *)layers[i].params;
      params[i] = malloc(sizeof(SoftmaxLayerParams));
      ((SoftmaxLayerParams *)params[i])
          ->initializeValues(user_params, this->data_type, this->tensor_format, batch_size,
                             current_output_size);
    }
  }

  h_layer_input = (void **)malloc((num_layers + 1) * sizeof(void *));  // host
  offloaded = (bool *)calloc((num_layers + 1), sizeof(bool));          // Offloaded layers index
}

void NeuralNet::allocate_mem_for_layers(std::vector<LayerSpecifier> &layers) {
  // Allocate space for parameters
  for (int i = 0; i < num_layers; i++) {
    size_t input_size;
    if (layers[i].type == CONV) {
      ConvDescriptor *user_params = (ConvDescriptor *)layers[i].params;
      ((ConvLayerParams *)params[i])
          ->allocateSpace(curand_gen, this->data_type, data_type_size, init_std_dev, free_bytes);
      input_size =
          batch_size * user_params->input_channels * user_params->input_h * user_params->input_w;
      if (i == 0) {
        input_channels = user_params->input_channels;
        input_h = user_params->input_h;
        input_w = user_params->input_w;
      }
    } else if (layers[i].type == FULLY_CONNECTED) {
      FCDescriptor *user_params = (FCDescriptor *)layers[i].params;
      ((FCLayerParams *)params[i])
          ->allocateSpace(curand_gen, this->data_type, data_type_size, init_std_dev, free_bytes);
      input_size = batch_size * user_params->input_channels;
      if (i == 0) {
        input_channels = user_params->input_channels;
        input_h = 1;
        input_w = 1;
      }
    } else if (layers[i].type == POOLING) {
      PoolingDescriptor *user_params = (PoolingDescriptor *)layers[i].params;
      ((PoolingLayerParams *)params[i])->allocateSpace(free_bytes);
      input_size =
          batch_size * user_params->input_channels * user_params->input_h * user_params->input_w;
      if (i == 0) {
        input_channels = user_params->input_channels;
        input_h = user_params->input_h;
        input_w = user_params->input_w;
      }
    } else if (layers[i].type == ACTV) {
      ActivationDescriptor *user_params = (ActivationDescriptor *)layers[i].params;
      ((ActivationLayerParams *)params[i])->allocateSpace(free_bytes);
      input_size = batch_size * user_params->channels * user_params->h * user_params->w;
      if (i == 0) {
        input_channels = user_params->channels;
        input_h = user_params->h;
        input_w = user_params->w;
      }
    } else if (layers[i].type == SOFTMAX) {
      SoftmaxDescriptor *user_params = (SoftmaxDescriptor *)layers[i].params;
      ((SoftmaxLayerParams *)params[i])->allocateSpace(free_bytes);
      input_size = batch_size * user_params->channels * user_params->h * user_params->w;

      layer_input_size[i + 1] = input_size;
      if (i == 0) {
        input_channels = user_params->channels;
        input_h = user_params->h;
        input_w = user_params->w;
      }
      if (i == num_layers - 1) num_classes = user_params->channels;
    }
    layer_input_size[i] = input_size;
  }
}

void NeuralNet::allocate_workspace_for_layers(std::vector<LayerSpecifier> &layers) {
  // Allocate space for workspace
  size_t cur_workspace_size_1, cur_workspace_size_2, cur_workspace_size_3, cur_workspace_size;
  this->workspace_size = 0;
  for (int i = 0; i < num_layers; i++) {
    if (layers[i].type == CONV) {
      cur_workspace_size_1 =
          ((ConvLayerParams *)params[i])->getWorkspaceSize(free_bytes, ConvLayerParams::FWD);
      cur_workspace_size_2 =
          ((ConvLayerParams *)params[i])->getWorkspaceSize(free_bytes, ConvLayerParams::BWD_DATA);
      cur_workspace_size_3 =
          ((ConvLayerParams *)params[i])->getWorkspaceSize(free_bytes, ConvLayerParams::BWD_FILTER);
      cur_workspace_size =
          max(cur_workspace_size_1, max(cur_workspace_size_2, cur_workspace_size_3));
      if (cur_workspace_size > workspace_size) this->workspace_size = cur_workspace_size;
    }
  }
}