#include <stdexcept>

#include "solver.cuh"

Solver::Solver(NeuralNet *model, void *X_train, int *y_train, void *X_val,
               int *y_val, int num_epoch, UpdateRule update_rule,
               double learning_rate, double learning_rate_decay, int num_train,
               int num_val)
{
  if ((model->batch_size == 0) || (model->num_layers == 0))
    throw std::invalid_argument(
        "Model is not initialized. Use parameterized constructor.");
  this->model = model;
  this->X_train = X_train, this->X_val = X_val;
  this->y_train = y_train, this->y_val = y_val;
  this->num_epoch = num_epoch;
  this->update_rule = update_rule;
  this->learning_rate = learning_rate,
  this->learning_rate_decay = learning_rate_decay;

  this->num_train = num_train, this->num_val = num_val;
  this->num_features = model->input_channels * model->input_h * model->input_w;

  hipEventCreate(&start);
  hipEventCreate(&stop);
}

float Solver::step(int start_X, int start_y, int *correct_count, bool train,
                   float *overhead, std::vector<std::pair<size_t, size_t>> &offload_mem)
{
  std::vector<float> t1, t2;
  return this->step(start_X, start_y, t1, t2, correct_count, train, overhead, offload_mem);
}

float Solver::step(int start_X, int start_y, std::vector<float> &fwd_dnn_lag,
                   std::vector<float> &bwd_dnn_lag, int *correct_count,
                   bool train, float *overhead, std::vector<std::pair<size_t, size_t>> &offload_mem)
{
  float temp_loss;

  if (model->data_type == HIPDNN_DATA_FLOAT)
    model->getLoss(&(((float *)X_train)[start_X]), &y_train[start_y],
                   learning_rate, fwd_dnn_lag, bwd_dnn_lag, offload_mem, train,
                   correct_count, &temp_loss, overhead);
  else if (model->data_type == HIPDNN_DATA_DOUBLE)
    model->getLoss(&(((double *)X_train)[start_X]), &y_train[start_y],
                   learning_rate, fwd_dnn_lag, bwd_dnn_lag, offload_mem, train,
                   correct_count, &temp_loss, overhead);

  hipDeviceSynchronize();
  return temp_loss;
}

void Solver::train(std::vector<float> &loss, std::vector<int> &val_acc,
                   std::vector<float> &batch_times, float *overhead, std::vector<std::pair<size_t, size_t>> &offload_mem)
{
  int batch_size = model->batch_size;
  int num_train_batches = num_train / model->batch_size;
  int num_val_batches = num_val / model->batch_size;

  for (int i = 0; i < num_epoch; i++)
  {
    // std::cout << "Epoch " << i << std::endl;
    for (int j = 0; j < num_train_batches; j++)
    {
      int start_sample = j * num_features * batch_size;

      float milli = 0;
      hipEventRecord(start, model->stream_compute);

      float temp_loss = step(start_sample, j * batch_size, NULL, true, overhead, offload_mem);

      hipEventRecord(stop, model->stream_compute);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milli, start, stop);

      loss.push_back(temp_loss);
      batch_times.push_back(milli);
    }

    int correct_count = 0;
    for (int j = 0; j < num_val_batches; j++)
    {
      int start_sample = j * num_features * batch_size;
      int temp_correct_count = 0;
      float temp_loss =
          step(start_sample, j * batch_size, &temp_correct_count, false, overhead, offload_mem);

      correct_count += temp_correct_count;
    }
    val_acc.push_back(correct_count);
    learning_rate *= learning_rate_decay;
  }
}

void Solver::checkAccuracy(void *X, int *y, int num_samples, int *num_correct)
{
  int batch_size = model->batch_size;
  int num_iter = num_samples / batch_size;
  *num_correct = 0;
  for (int i = 0; i < num_iter; i++)
  {
    int start_sample = i * num_features * batch_size;
    int temp_correct_count;
    std::vector<std::pair<size_t, size_t>> offload_mem;
    if (model->data_type == HIPDNN_DATA_FLOAT)
      model->getLoss(&(((float *)X)[start_sample]), &y[i * batch_size],
                     learning_rate, offload_mem, false, &temp_correct_count, NULL, NULL);
    else if (model->data_type == HIPDNN_DATA_DOUBLE)
      model->getLoss(&(((double *)X)[start_sample]), &y[i * batch_size],
                     learning_rate, offload_mem, false, &temp_correct_count, NULL, NULL);
    *num_correct = *num_correct + temp_correct_count;
  }
}
