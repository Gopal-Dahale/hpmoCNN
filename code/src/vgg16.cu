#include "hip/hip_runtime.h"
/**
 * Upsampled MNIST Dataset
 * VGG Net
 */

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <set>
#include <string>
#include <unordered_map>
#include <vector>

#include "cxxopts.hpp"
#include "solver.cuh"

using namespace std;

typedef unsigned char uchar;

int num_train = 1024, num_test = 512;

void save_offload_mem(vector<pair<size_t, size_t>> &offload_mem, int nn, int batch_size) {
  fstream f;
  string res = "vgg" + to_string(nn) + "/batch_size" + to_string(batch_size);
  f.open("../" + res + "/offload_mem.txt", ios::out);
  for (auto &i : offload_mem) f << i.first << " " << i.second << endl;
  f.close();
}

void save_metrics(vector<float> &loss, vector<int> &val_acc, vector<float> &batch_times,
                  unordered_map<string, double> &configs, float total_train_time, float overhead,
                  int nn, int batch_size) {
  fstream f;

  string res = "vgg" + to_string(nn) + "/batch_size" + to_string(batch_size);
  f.open("../" + res + "/batch_times.txt", ios::out);
  for (auto &i : batch_times) f << i << endl;
  f.close();

  f.open("../" + res + "/loss.txt", ios::out);
  for (auto &i : loss) f << i << endl;
  f.close();

  f.open("../" + res + "/val_acc.txt", ios::out);
  for (auto &i : val_acc) f << i << endl;
  f.close();

  f.open("../" + res + "/configs.txt", ios::out);
  for (auto &i : configs) f << i.first << " " << i.second << endl;
  f.close();

  f.open("../" + res + "/totaltime.txt", ios::out);
  f << total_train_time << endl;
  f.close();

  f.open("../" + res + "/totaloverhead.txt", ios::out);
  f << overhead << endl;
  f.close();
}

void save_mem_usage(NeuralNet *net, int nn, int batch_size) {
  std::ofstream mem_usage;
  string res = "vgg" + to_string(nn) + "/batch_size" + to_string(batch_size);
  mem_usage.open("../" + res + "/mem_usage.txt");

  for (int c = 0; c < net->num_layers + 1; c++) {
    size_t feature_map_size, fwd_workspace_size = 0, bwd_workspace_filter = 0,
                             bwd_workspace_data = 0, weights = 0;
    feature_map_size = net->layer_input_size[c] * net->data_type_size;
    if (c != net->num_layers && net->layer_type[c] == CONV) {
      ConvLayerParams *cur_params = (ConvLayerParams *)net->params[c];
      fwd_workspace_size = cur_params->fwd_workspace_size;
      bwd_workspace_filter = cur_params->bwd_filter_workspace_size;
      bwd_workspace_data = cur_params->bwd_data_workspace_size;
      weights = cur_params->kernel_size * net->data_type_size;
    } else if (c != net->num_layers && net->layer_type[c] == FULLY_CONNECTED) {
      FCLayerParams *cur_params = (FCLayerParams *)net->params[c];
      int wt_alloc_size = cur_params->weight_matrix_size;
      if (wt_alloc_size % 2 != 0) wt_alloc_size += 1;
      weights = (wt_alloc_size + cur_params->C_out) * net->data_type_size;
    }
    mem_usage << feature_map_size << " " << fwd_workspace_size << " " << bwd_workspace_filter << " "
              << bwd_workspace_data << " " << weights << "\n";
    // std::cout << feature_map_size << " " << fwd_workspace_size << " " << bwd_workspace_filter <<
    // " " << bwd_workspace_data << " " << weights << "\n"; total_feature_map_size +=
    // layer_input_size[c] * data_type_size;
  }
  mem_usage.close();
}

int reverseInt(int n) {
  const int bytes = 4;
  unsigned char ch[bytes];
  for (int i = 0; i < bytes; i++) {
    ch[i] = (n >> i * 8) & 255;
  }
  int p = 0;
  for (int i = 0; i < bytes; i++) {
    p += (int)ch[i] << (bytes - i - 1) * 8;
  }
  return p;
}

void readMNIST224(vector<vector<uchar>> &train_images, vector<vector<uchar>> &test_images,
                  vector<uchar> &train_labels, vector<uchar> &test_labels, int num_train,
                  int num_test) {
  string filename_train_images = "/kaggle/input/mnist224by224testdataset/train-images-224by224-";
  string filename_train_labels = "data/train-labels.idx1-ubyte";

  string filename_test_images = "/kaggle/input/mnist224by224testdataset/test-images-224by224-";
  string filename_test_labels = "data/t10k-labels.idx1-ubyte";

  // read train/test images
  int images_per_file = 2000;
  int num_train_files = min((int)(ceil(num_train / float(images_per_file))), 30);
  int num_test_files = min((int)(ceil(num_test / float(images_per_file))), 5);

  for (int i = 0; i < 2; i++) {
    int num_files = (i == 0 ? num_train_files : num_test_files);
    for (int j = 0; j < num_files; j++) {
      string filename;
      if (i == 0)
        filename = filename_train_images;
      else
        filename = filename_test_images;
      filename = filename + to_string(j) + ".idx3-ubyte";

      ifstream f(filename.c_str(), ios::binary);
      if (!f.is_open()) printf("Cannot read MNIST from %s\n", filename.c_str());

      // read metadata
      int magic_number = 0, n_images = 0, n_rows = 0, n_cols = 0;
      f.read((char *)&magic_number, sizeof(magic_number));
      magic_number = reverseInt(magic_number);
      f.read((char *)&n_images, sizeof(n_images));
      n_images = reverseInt(n_images);
      f.read((char *)&n_rows, sizeof(n_rows));
      n_rows = reverseInt(n_rows);
      f.read((char *)&n_cols, sizeof(n_cols));
      n_cols = reverseInt(n_cols);

      for (int k = 0; k < n_images; k++) {
        vector<uchar> temp;
        temp.reserve(n_rows * n_cols);
        for (int j = 0; j < n_rows * n_cols; j++) {
          uchar t = 0;
          f.read((char *)&t, sizeof(t));
          temp.push_back(t);
        }
        if (i == 0) {
          train_images.push_back(temp);
          if ((j * n_images + k + 1) >= num_train) break;
        } else {
          test_images.push_back(temp);
          if ((j * n_images + k + 1) >= num_test) break;
        }
      }
      f.close();
    }
  }

  // read train/test labels
  for (int i = 0; i < 2; i++) {
    string filename;
    if (i == 0)
      filename = filename_train_labels;
    else
      filename = filename_test_labels;

    ifstream f(filename.c_str(), ios::binary);
    if (!f.is_open()) printf("Cannot read MNIST from %s\n", filename.c_str());

    // read metadata
    int magic_number = 0, n_labels = 0;
    f.read((char *)&magic_number, sizeof(magic_number));
    magic_number = reverseInt(magic_number);
    f.read((char *)&n_labels, sizeof(n_labels));
    n_labels = reverseInt(n_labels);

    if (i == 0)
      n_labels = min(n_labels, num_train);
    else
      n_labels = min(n_labels, num_test);

    for (int k = 0; k < n_labels; k++) {
      uchar t = 0;
      f.read((char *)&t, sizeof(t));
      if (i == 0)
        train_labels.push_back(t);
      else
        test_labels.push_back(t);
    }
    f.close();
  }

  assert(train_images.size() == train_labels.size());
  assert(test_images.size() == test_labels.size());
}

int main(int argc, char *argv[]) {
  /******************* Parse command line arguments ********************/
  cxxopts::Options options("hpmoCNN",
                           "High Performance Memory Optimal Convolutional Neural Network");

  options.add_options()(
      "batch-size", "Batch Size",
      cxxopts::value<int>()->default_value("64")) // a bool parameter
      ("softmax-eps", "softmax eps",
       cxxopts::value<float>()->default_value("1e-8"))(
          "init-std-dev", "initial standard deviation",
          cxxopts::value<float>()->default_value("0.01"))(
          "epochs", "Number of epochs",
          cxxopts::value<int>()->default_value("5"))(
          "learning-rate", "Learning Rate",
          cxxopts::value<double>()->default_value("0.01"))(
          "learning-rate-decay", "Learning Rate Decay",
          cxxopts::value<double>()->default_value("1"))(
          "num-train", "Number of training examples to use",
          cxxopts::value<int>()->default_value("1024"))(
          "num-test", "Number of testing examples to use",
          cxxopts::value<int>()->default_value("512"))(
          "nn", "neural network",
          cxxopts::value<int>()->default_value("16"))("help", "Print Usage");

  auto result = options.parse(argc, argv);
  if (result.count("help")) {
    std::cout << options.help() << std::endl;
    exit(0);
  }

  num_train = result["num-train"].as<int>();
  num_test = result["num-test"].as<int>();

  /******************* Read Dataset ************************************/

  bool doo = false;
  int rows = 224, cols = 224, channels = 1;
  vector<vector<uchar>> train_images, test_images;
  vector<uchar> train_labels, test_labels;

  cout << "Reading MNIST dataset...  ";

  readMNIST224(train_images, test_images, train_labels, test_labels, num_train, num_test);

  assert(train_images.size() == train_labels.size());
  assert(test_images.size() == test_labels.size());

  float *f_train_images, *f_test_images;
  int *f_train_labels, *f_test_labels;

  int input_size = rows * cols * channels;
  f_train_images = (float *)malloc(num_train * input_size * sizeof(float));
  f_train_labels = (int *)malloc(num_train * sizeof(int));
  f_test_images = (float *)malloc(num_test * input_size * sizeof(float));
  f_test_labels = (int *)malloc(num_test * sizeof(int));

  for (int k = 0; k < num_train; k++) {
    for (int j = 0; j < input_size; j++)
      f_train_images[k * input_size + j] = (float)train_images[k][j];
    f_train_labels[k] = (int)train_labels[k];
  }

  for (int k = 0; k < num_test; k++) {
    for (int j = 0; j < input_size; j++)
      f_test_images[k * input_size + j] = (float)test_images[k][j];
    f_test_labels[k] = (int)test_labels[k];
  }

  float *mean_image;
  mean_image = (float *)malloc(input_size * sizeof(float));

  for (int i = 0; i < input_size; i++) {
    mean_image[i] = 0;
    for (int k = 0; k < num_train; k++) mean_image[i] += f_train_images[k * input_size + i];
    mean_image[i] /= num_train;
  }

  for (int i = 0; i < num_train; i++) {
    for (int j = 0; j < input_size; j++) f_train_images[i * input_size + j] -= mean_image[j];
  }

  for (int i = 0; i < input_size; i++) {
    mean_image[i] = 0;
    for (int k = 0; k < num_test; k++) mean_image[i] += f_test_images[k * input_size + i];
    mean_image[i] /= num_test;
  }

  for (int i = 0; i < num_test; i++) {
    for (int j = 0; j < input_size; j++) f_test_images[i * input_size + j] -= mean_image[j];
  }

  cout << "Done" << endl;

  /******************* VGG NET ************************************/
  vector<LayerSpecifier> layer_specifier;
  {
    ConvDescriptor part0_conv0;
    part0_conv0.initializeValues(1, 64, 3, 3, 224, 224, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part0_conv0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part0_conv1;
    part0_conv1.initializeValues(64, 64, 3, 3, 224, 224, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part0_conv1;
    layer_specifier.push_back(temp);
  }
  {
    PoolingDescriptor pool0;
    pool0.initializeValues(64, 2, 2, 224, 224, 0, 0, 2, 2, POOLING_MAX);
    LayerSpecifier temp;
    temp.initPointer(POOLING);
    *((PoolingDescriptor *)temp.params) = pool0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part1_conv0;
    part1_conv0.initializeValues(64, 128, 3, 3, 112, 112, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part1_conv0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part1_conv1;
    part1_conv1.initializeValues(128, 128, 3, 3, 112, 112, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part1_conv1;
    layer_specifier.push_back(temp);
  }
  {
    PoolingDescriptor pool1;
    pool1.initializeValues(128, 2, 2, 112, 112, 0, 0, 2, 2, POOLING_MAX);
    LayerSpecifier temp;
    temp.initPointer(POOLING);
    *((PoolingDescriptor *)temp.params) = pool1;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part2_conv0;
    part2_conv0.initializeValues(128, 256, 3, 3, 56, 56, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part2_conv0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part2_conv1;
    part2_conv1.initializeValues(256, 256, 3, 3, 56, 56, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part2_conv1;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part2_conv2;
    part2_conv2.initializeValues(256, 256, 3, 3, 56, 56, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part2_conv2;
    layer_specifier.push_back(temp);
  }
  {
    PoolingDescriptor pool2;
    pool2.initializeValues(256, 2, 2, 56, 56, 0, 0, 2, 2, POOLING_MAX);
    LayerSpecifier temp;
    temp.initPointer(POOLING);
    *((PoolingDescriptor *)temp.params) = pool2;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part3_conv0;
    part3_conv0.initializeValues(256, 512, 3, 3, 28, 28, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part3_conv0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part3_conv1;
    part3_conv1.initializeValues(512, 512, 3, 3, 28, 28, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part3_conv1;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part3_conv2;
    part3_conv2.initializeValues(512, 512, 3, 3, 28, 28, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part3_conv2;
    layer_specifier.push_back(temp);
  }
  {
    PoolingDescriptor pool3;
    pool3.initializeValues(512, 2, 2, 28, 28, 0, 0, 2, 2, POOLING_MAX);
    LayerSpecifier temp;
    temp.initPointer(POOLING);
    *((PoolingDescriptor *)temp.params) = pool3;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part4_conv0;
    part4_conv0.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part4_conv0;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part4_conv1;
    part4_conv1.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part4_conv1;
    layer_specifier.push_back(temp);
  }
  {
    ConvDescriptor part4_conv2;
    part4_conv2.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1, RELU);
    LayerSpecifier temp;
    temp.initPointer(CONV);
    *((ConvDescriptor *)temp.params) = part4_conv2;
    layer_specifier.push_back(temp);
  }
  {
    PoolingDescriptor pool3;
    pool3.initializeValues(512, 2, 2, 14, 14, 0, 0, 2, 2, POOLING_MAX);
    LayerSpecifier temp;
    temp.initPointer(POOLING);
    *((PoolingDescriptor *)temp.params) = pool3;
    layer_specifier.push_back(temp);
  }

  {
    FCDescriptor part5_fc0;
    part5_fc0.initializeValues(7 * 7 * 512, 4096, RELU);
    LayerSpecifier temp;
    temp.initPointer(FULLY_CONNECTED);
    *((FCDescriptor *)temp.params) = part5_fc0;
    layer_specifier.push_back(temp);
  }
  {
    FCDescriptor part5_fc1;
    part5_fc1.initializeValues(4096, 4096, RELU);
    LayerSpecifier temp;
    temp.initPointer(FULLY_CONNECTED);
    *((FCDescriptor *)temp.params) = part5_fc1;
    layer_specifier.push_back(temp);
  }
  {
    FCDescriptor part5_fc2;
    part5_fc2.initializeValues(4096, 1000, RELU);
    LayerSpecifier temp;
    temp.initPointer(FULLY_CONNECTED);
    *((FCDescriptor *)temp.params) = part5_fc2;
    layer_specifier.push_back(temp);
  }
  {
    FCDescriptor part5_fc3;
    part5_fc3.initializeValues(1000, 100, RELU);
    LayerSpecifier temp;
    temp.initPointer(FULLY_CONNECTED);
    *((FCDescriptor *)temp.params) = part5_fc3;
    layer_specifier.push_back(temp);
  }
  {
    FCDescriptor part5_fc3;
    part5_fc3.initializeValues(100, 10, RELU);
    LayerSpecifier temp;
    temp.initPointer(FULLY_CONNECTED);
    *((FCDescriptor *)temp.params) = part5_fc3;
    layer_specifier.push_back(temp);
  }
  {
    SoftmaxDescriptor s_max;
    s_max.initializeValues(SOFTMAX_ACCURATE, SOFTMAX_MODE_INSTANCE, 10, 1, 1);
    LayerSpecifier temp;
    temp.initPointer(SOFTMAX);
    *((SoftmaxDescriptor *)temp.params) = s_max;
    layer_specifier.push_back(temp);
  }

  /**************************** Configuration ****************************/
  int batch_size = result["batch-size"].as<int>();
  float softmax_eps = result["softmax-eps"].as<float>();
  float init_std_dev = result["init-std-dev"].as<float>();
  int num_epoch = result["epochs"].as<int>();
  double learning_rate = result["learning-rate"].as<double>();
  double learning_rate_decay = result["learning-rate-decay"].as<double>();
  int nn = result["nn"].as<int>();

  /************************ Display configuration *************************/
  unordered_map<string, double> configs = {{"neural_network: vgg", nn},
                                           {"batch_size", batch_size},
                                           {"softmax_eps", softmax_eps},
                                           {"init_std_dev", init_std_dev},
                                           {"num_epoch", num_epoch},
                                           {"learning_rate", learning_rate},
                                           {"learning_rate_decay", learning_rate_decay},
                                           {"num_train", num_train},
                                           {"num_test", num_test}};

  for (auto &config : configs) {
    cout << config.first << ": " << config.second << endl;
  }

  /*************************** Train & Test ***************************/
  NeuralNet net(layer_specifier, DATA_FLOAT, batch_size, TENSOR_NCHW, softmax_eps, init_std_dev,
                SGD);
  Solver solver(&net, (void *)f_train_images, f_train_labels, (void *)f_train_images,
                f_train_labels, num_epoch, SGD, learning_rate, learning_rate_decay, num_train,
                num_train);
  vector<float> loss;
  vector<int> val_acc;
  vector<float> batch_times;
  vector<pair<size_t, size_t>> offload_mem;
  float milli = 0, overhead = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  solver.train(loss, val_acc, batch_times, &overhead, offload_mem);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  int num_correct;

  solver.checkAccuracy(f_train_images, f_train_labels, num_train, &num_correct);
  std::cout << "TRAIN NUM CORRECT:" << num_correct << endl;
  solver.checkAccuracy(f_test_images, f_test_labels, num_test, &num_correct);
  std::cout << "TEST NUM CORRECT:" << num_correct << endl;

  /*************************** Save metrics ***************************/
  save_mem_usage(&net, nn, batch_size);
  save_metrics(loss, val_acc, batch_times, configs, milli, overhead, nn, batch_size);
  save_offload_mem(offload_mem, nn, batch_size);
}
